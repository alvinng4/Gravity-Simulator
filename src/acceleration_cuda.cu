#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "acceleration_cuda.cuh"

WIN32DLL_API void acceleration_pairwise_cuda(
    int objects_count,
    double *__restrict x,
    double *__restrict a,
    const double *__restrict m,
    double G,
    double softening_length
)
{
    double *x_device = NULL;
    double *a_device = NULL;
    double *m_device = NULL;
    hipError_t error = hipSuccess;

    error = hipMalloc((double **) &x_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &a_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &m_device, objects_count * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }

    error = hipMemcpy(x_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }
    error = hipMemcpy(m_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }

    acceleration_pairwise_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        objects_count,
        x_device,
        a_device,
        m_device,
        G,
        softening_length
    );

    error = hipMemcpy(a, a_device, objects_count * sizeof(double3), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from device to host\n");
        goto err_gpu_memory;
    }

    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
    return;

err_gpu_memory:
    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
}

__global__ void acceleration_pairwise_kernel(
    int objects_count,
    double *__restrict x,
    double *__restrict a,
    const double *__restrict m,
    double G,
    double softening_length
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < objects_count)
    {
        double ax = 0.0;
        double ay = 0.0;
        double az = 0.0;
        double obj_i_x = x[i * 3 + 0];
        double obj_i_y = x[i * 3 + 1];
        double obj_i_z = x[i * 3 + 2];

        for (int j = 0; j < objects_count; j++)
        {
            if (i != j)
            {
                double dx = x[j * 3 + 0] - obj_i_x;
                double dy = x[j * 3 + 1] - obj_i_y;
                double dz = x[j * 3 + 2] - obj_i_z;
                double r_norm = sqrt(dx * dx + dy * dy + dz * dz + softening_length * softening_length);

                double temp_value = G * m[j] / (r_norm * r_norm * r_norm);
                ax += temp_value * dx;
                ay += temp_value * dy;
                az += temp_value * dz;
            }
        }
        a[i * 3 + 0] = ax;
        a[i * 3 + 1] = ay;
        a[i * 3 + 2] = az;
    }
}

WIN32DLL_API void acceleration_pairwise_float_cuda(
    int objects_count,
    double *__restrict x,
    double *__restrict a,
    const double *__restrict m,
    double G,
    double softening_length
)
{
    float *x_device = NULL;
    float *a_device = NULL;
    float *m_device = NULL;
    float *x_float = NULL;
    float *a_float = NULL;
    float *m_float = NULL;
    hipError_t error = hipSuccess;

    error = hipMalloc((float **) &x_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &a_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &m_device, objects_count * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }

    x_float = (float *) malloc(objects_count * 3 * sizeof(float));
    a_float = (float *) malloc(objects_count * 3 * sizeof(float));
    m_float = (float *) malloc(objects_count * sizeof(float));
    if (x_float == NULL || a_float == NULL || m_float == NULL)
    {
        fprintf(stderr, "Error: Failed to allocate cpu memory for calculation\n");
        goto err_cpu_memory;
    }

    for (int i = 0; i < objects_count; i++)
    {
        x_float[i * 3 + 0] = (float) x[i * 3 + 0];
        x_float[i * 3 + 1] = (float) x[i * 3 + 1];
        x_float[i * 3 + 2] = (float) x[i * 3 + 2];
        m_float[i] = (float) m[i];
    }

    error = hipMemcpy(x_device, x_float, objects_count * 3 * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }
    error = hipMemcpy(m_device, m_float, objects_count * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }

    acceleration_pairwise_float_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        objects_count,
        x_device,
        a_device,
        m_device,
        G,
        softening_length
    );

    error = hipMemcpy(a_float, a_device, objects_count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from device to host\n");
        goto err_gpu_memory;
    }
    for (int i = 0; i < objects_count; i++)
    {
        a[i * 3 + 0] = (double) a_float[i * 3 + 0];
        a[i * 3 + 1] = (double) a_float[i * 3 + 1];
        a[i * 3 + 2] = (double) a_float[i * 3 + 2];
    }

    free(x_float);
    free(a_float);
    free(m_float);
    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
    return;

err_cpu_memory:
    free(x_float);
    free(a_float);
    free(m_float);
err_gpu_memory:
    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
}

__global__ void acceleration_pairwise_float_kernel(
    int objects_count,
    float *__restrict x,
    float *__restrict a,
    const float *__restrict m,
    float G,
    float softening_length
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < objects_count)
    {
        float ax = 0.0;
        float ay = 0.0;
        float az = 0.0;
        float obj_i_x = x[i * 3 + 0];
        float obj_i_y = x[i * 3 + 1];
        float obj_i_z = x[i * 3 + 2];

        for (int j = 0; j < objects_count; j++)
        {
            if (i != j)
            {
                float dx = x[j * 3 + 0] - obj_i_x;
                float dy = x[j * 3 + 1] - obj_i_y;
                float dz = x[j * 3 + 2] - obj_i_z;
                float r_norm = sqrtf(dx * dx + dy * dy + dz * dz + softening_length * softening_length);

                float temp_value = G * m[j] / (r_norm * r_norm * r_norm);
                ax += temp_value * dx;
                ay += temp_value * dy;
                az += temp_value * dz;
            }
        }
        a[i * 3 + 0] = ax;
        a[i * 3 + 1] = ay;
        a[i * 3 + 2] = az;
    }
}

WIN32DLL_API void acceleration_pairwise_float_comp_sum_cuda(
    int objects_count,
    double *__restrict x,
    double *__restrict a,
    const double *__restrict m,
    double G,
    double softening_length
)
{
    float *x_device = NULL;
    float *a_device = NULL;
    float *m_device = NULL;
    float *x_float = NULL;
    float *a_float = NULL;
    float *m_float = NULL;
    hipError_t error = hipSuccess;

    error = hipMalloc((float **) &x_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &a_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &m_device, objects_count * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate gpu memory for calculation\n");
        goto err_gpu_memory;
    }

    x_float = (float *) malloc(objects_count * 3 * sizeof(float));
    a_float = (float *) malloc(objects_count * 3 * sizeof(float));
    m_float = (float *) malloc(objects_count * sizeof(float));
    if (x_float == NULL || a_float == NULL || m_float == NULL)
    {
        fprintf(stderr, "Error: Failed to allocate cpu memory for calculation\n");
        goto err_cpu_memory;
    }

    for (int i = 0; i < objects_count; i++)
    {
        x_float[i * 3 + 0] = (float) x[i * 3 + 0];
        x_float[i * 3 + 1] = (float) x[i * 3 + 1];
        x_float[i * 3 + 2] = (float) x[i * 3 + 2];
        m_float[i] = (float) m[i];
    }

    error = hipMemcpy(x_device, x_float, objects_count * 3 * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }
    error = hipMemcpy(m_device, m_float, objects_count * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to device\n");
        goto err_gpu_memory;
    }

    acceleration_pairwise_float_comp_sum_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        objects_count,
        x_device,
        a_device,
        m_device,
        G,
        softening_length
    );

    error = hipMemcpy(a_float, a_device, objects_count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from device to host\n");
        goto err_gpu_memory;
    }
    for (int i = 0; i < objects_count; i++)
    {
        a[i * 3 + 0] = (double) a_float[i * 3 + 0];
        a[i * 3 + 1] = (double) a_float[i * 3 + 1];
        a[i * 3 + 2] = (double) a_float[i * 3 + 2];
    }

    free(x_float);
    free(a_float);
    free(m_float);
    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
    return;

err_cpu_memory:
    free(x_float);
    free(a_float);
    free(m_float);
err_gpu_memory:
    hipFree(x_device);
    hipFree(a_device);
    hipFree(m_device);
}

__global__ void acceleration_pairwise_float_comp_sum_kernel(
    int objects_count,
    float *__restrict x,
    float *__restrict a,
    const float *__restrict m,
    float G,
    float softening_length
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < objects_count)
    {
        float ax = 0.0;
        float ay = 0.0;
        float az = 0.0;
        float obj_i_x = x[i * 3 + 0];
        float obj_i_y = x[i * 3 + 1];
        float obj_i_z = x[i * 3 + 2];
        float ax0 = 0.0;
        float ay0 = 0.0;
        float az0 = 0.0;
        float x_err_comp_sum = 0.0;
        float y_err_comp_sum = 0.0;
        float z_err_comp_sum = 0.0;

        for (int j = 0; j < objects_count; j++)
        {
            if (i != j)
            {
                float dx = x[j * 3 + 0] - obj_i_x;
                float dy = x[j * 3 + 1] - obj_i_y;
                float dz = x[j * 3 + 2] - obj_i_z;
                float r_norm = sqrtf(dx * dx + dy * dy + dz * dz + softening_length * softening_length);

                float temp_value = G * m[j] / (r_norm * r_norm * r_norm);

                ax0 = ax;
                ay0 = ay;
                az0 = az;

                x_err_comp_sum += temp_value * dx;
                y_err_comp_sum += temp_value * dy;
                z_err_comp_sum += temp_value * dz;

                ax = ax0 + x_err_comp_sum;
                ay = ay0 + y_err_comp_sum;
                az = az0 + z_err_comp_sum;

                x_err_comp_sum += ax0 - ax;
                y_err_comp_sum += ay0 - ay;
                z_err_comp_sum += az0 - az;
            }
        }
        a[i * 3 + 0] = ax;
        a[i * 3 + 1] = ay;
        a[i * 3 + 2] = az;
    }
}