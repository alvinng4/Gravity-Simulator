#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "acceleration_cuda.cuh"
#include "error.h"
#include "gravity_sim.h"

__global__ void acceleration_pairwise_kernel(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    double3 local_a = make_double3(0.0, 0.0, 0.0);
    const double3 x_i = make_double3(
        x[i * 3 + 0],
        x[i * 3 + 1],
        x[i * 3 + 2]
    );

    for (int j = 0; j < objects_count; j++)
    {
        if (i == j)
        {
            continue;
        }

        double3 dx = make_double3(
            x_i.x - x[j * 3 + 0],
            x_i.y - x[j * 3 + 1],
            x_i.z - x[j * 3 + 2]
        );
        const double r_norm = sqrt(
            dx.x * dx.x + dx.y * dx.y + dx.z * dx.z + softening_length * softening_length
        );
        const double temp_value = G * m[j] / (r_norm * r_norm * r_norm);
        local_a.x -= temp_value * dx.x;
        local_a.y -= temp_value * dx.y;
        local_a.z -= temp_value * dx.z;
    }
    a[i * 3 + 0] = local_a.x;
    a[i * 3 + 1] = local_a.y;
    a[i * 3 + 2] = local_a.z;
}

extern "C"
{
    WIN32DLL_API int acceleration_pairwise_cuda(
        double *__restrict a,
        const System *__restrict system,
        const AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;

        const int objects_count = system->objects_count;
        const double *x = system->x;
        const double *m = system->m;
        const double G = system->G;
        const double softening_length = acceleration_param->softening_length;

        double *a_device = NULL;
        double *x_device = NULL;
        double *m_device = NULL;
        hipError_t error;

        error = hipMalloc((double **) &a_device, objects_count * 3 * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((double **) &x_device, objects_count * 3 * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((double **) &m_device, objects_count * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }

        error = hipMemcpy(x_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_CPU_TO_GPU;
            goto err_memcpy_cpu_to_gpu;
        }
        error = hipMemcpy(m_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_CPU_TO_GPU;
            goto err_memcpy_cpu_to_gpu;
        }

        acceleration_pairwise_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
            a_device,
            objects_count,
            x_device,
            m_device,
            G,
            softening_length
        );

        error = hipMemcpy(a, a_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_GPU_TO_CPU;
            goto err_memcpy_gpu_to_cpu;
        }

        hipFree(a_device);
        hipFree(x_device);
        hipFree(m_device);

        return SUCCESS;

    err_memcpy_gpu_to_cpu:
    err_memcpy_cpu_to_gpu:
    err_gpu_memory:
        hipFree(a_device);
        hipFree(x_device);
        hipFree(m_device);
        return return_code;
    }
}

__global__ void memcpy_array_double_to_float(
    const double *__restrict x_double,
    const double *__restrict m_double,
    const int objects_count,
    float *__restrict x,
    float *__restrict m
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    x[i * 3 + 0] = x_double[i * 3 + 0];
    x[i * 3 + 1] = x_double[i * 3 + 1];
    x[i * 3 + 2] = x_double[i * 3 + 2];
    m[i] = m_double[i];

    return;
}

__global__ void acceleration_pairwise_float_kernel(
    double *__restrict a_double,
    const int objects_count,
    const float *__restrict x,
    const float *__restrict m,
    const float G,
    const float softening_length
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    float3 local_a = make_float3(0.0, 0.0, 0.0);
    const float3 x_i = make_float3(
        x[i * 3 + 0],
        x[i * 3 + 1],
        x[i * 3 + 2]
    );

    for (int j = 0; j < objects_count; j++)
    {
        if (i == j)
        {
            continue;
        }

        float3 dx = make_float3(
            x_i.x - x[j * 3 + 0],
            x_i.y - x[j * 3 + 1],
            x_i.z - x[j * 3 + 2]
        );
        const float r_norm = sqrt(
            dx.x * dx.x + dx.y * dx.y + dx.z * dx.z + softening_length * softening_length
        );
        const float temp_value = G * m[j] / (r_norm * r_norm * r_norm);
        local_a.x -= temp_value * dx.x;
        local_a.y -= temp_value * dx.y;
        local_a.z -= temp_value * dx.z;
    }
    a_double[i * 3 + 0] = local_a.x;
    a_double[i * 3 + 1] = local_a.y;
    a_double[i * 3 + 2] = local_a.z;
}

extern "C"
{
    WIN32DLL_API int acceleration_pairwise_cuda_float(
        double *__restrict a,
        const System *__restrict system,
        const AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;

        const int objects_count = system->objects_count;
        const double *x = system->x;
        const double *m = system->m;
        const double G = system->G;
        const double softening_length = acceleration_param->softening_length;

        double *a_double_device = NULL;
        double *x_double_device = NULL;
        double *m_double_device = NULL;
        float *x_device = NULL;
        float *m_device = NULL;
        hipError_t error;

        error = hipMalloc((double **) &a_double_device, objects_count * 3 * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((double **) &x_double_device, objects_count * 3 * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((double **) &m_double_device, objects_count * sizeof(double));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((float **) &x_device, objects_count * 3 * sizeof(float));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }
        error = hipMalloc((float **) &m_device, objects_count * sizeof(float));
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMORY_ALLOC;
            goto err_gpu_memory;
        }

        error = hipMemcpy(x_double_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_CPU_TO_GPU;
            goto err_memcpy_cpu_to_gpu;
        }
        error = hipMemcpy(m_double_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_CPU_TO_GPU;
            goto err_memcpy_cpu_to_gpu;
        }

        memcpy_array_double_to_float <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
            x_double_device,
            m_double_device,
            objects_count,
            x_device,
            m_device
        );

        acceleration_pairwise_float_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
            a_double_device,
            objects_count,
            x_device,
            m_device,
            G,
            softening_length
        );

        error = hipMemcpy(a, a_double_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            return_code = ERROR_CUDA_PAIRWISE_MEMCPY_GPU_TO_CPU;
            goto err_memcpy_gpu_to_cpu;
        }

        hipFree(a_double_device);
        hipFree(x_double_device);
        hipFree(m_double_device);
        hipFree(x_device);
        hipFree(m_device);

        return SUCCESS;

    err_memcpy_gpu_to_cpu:
    err_memcpy_cpu_to_gpu:
    err_gpu_memory:
        hipFree(a_double_device);
        hipFree(x_double_device);
        hipFree(m_double_device);
        hipFree(x_device);
        hipFree(m_device);
        return return_code;
    }
}
