#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "acceleration_barnes_hut.h"
#include "acceleration_cuda.cuh"
#include "error.h"
#include "gravity_sim.h"
#include "math_functions.h"


/**
 * \brief Check if two Morton indices are included in the same octant
 * 
 * \param morton_index_i Morton index of the first object at the deepest level
 * \param morton_index_j Morton index of the second object at the deepest level
 * \param level Level of the Morton indices
 */
 inline __device__ bool _check_if_included(
    const int64 morton_index_i,
    const int64 morton_index_j,
    const int level
)
{
    return (morton_index_i >> (3 * (MORTON_MAX_LEVEL - level))) == (morton_index_j >> (3 * (MORTON_MAX_LEVEL - level)));
}

/**
 * \brief Helper kernel function for computing the acceleration of one particle
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
__global__ void _acceleration_helper_kernel(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    typedef struct Stack
    {
        int node;
        int processed_children;
        struct Stack *last;
    } Stack;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    const int idx_i = sorted_indices[i];
    const int64 morton_index_i = leaf_morton_indices_deepest_level[idx_i];
    const double3 x_i = make_double3(x[idx_i * 3 + 0], x[idx_i * 3 + 1], x[idx_i * 3 + 2]);
    const double softening_squared = softening_length * softening_length;
    
    Stack stack_pool[MORTON_MAX_LEVEL];
    Stack *stack = &(stack_pool[0]);
    stack->processed_children = -1;
    stack->last = NULL;
    stack->node = 0;
    double3 local_a = make_double3(0.0, 0.0, 0.0);

    int level = 1;

    /* Tree walk */
    while (true)
    {
        const int node = stack->node;
        for (int j = (stack->processed_children) + 1; j < tree_num_internal_children[node]; j++)
        {
            const int child_j = tree_idx_first_internal_child[node] + j;
            const int num_children_j = tree_num_internal_children[child_j];
            const int start_idx_j = tree_start_particle_sorted_idx[child_j];

            /* Leaf node */
            if (num_children_j <= 0)
            {
                const int num_particles_j = tree_num_particles[child_j];
                for (int k = 0; k < num_particles_j; k++)
                {
                    const int idx_j = sorted_indices[start_idx_j + k];
                    if (idx_i == idx_j)
                    {
                        continue;
                    }

                    // Calculate \vec{R} and its norm
                    const double3 dr = make_double3(
                        x_i.x - x[idx_j * 3 + 0],
                        x_i.y - x[idx_j * 3 + 1],
                        x_i.z - x[idx_j * 3 + 2]
                    );
                    const double R_norm = sqrt(
                        dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + softening_squared
                    );

                    // Calculate the acceleration
                    const double temp_value = G * m[idx_j] / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x;
                    local_a.y -= temp_value * dr.y;
                    local_a.z -= temp_value * dr.z;
                }

                stack->processed_children = j;
                continue;
            }

            /* Internal node */
            else
            {
                bool criteria_met = false;

                // If object i is included, then we need to traverse deeper
                const bool is_included = _check_if_included(
                    morton_index_i,
                    leaf_morton_indices_deepest_level[sorted_indices[start_idx_j]],
                    level
                );

                // Check Barnes-Hut criteria
                double3 dr;
                double norm_square;
                if (!is_included)
                {
                    dr = make_double3(
                        x_i.x - tree_center_of_mass_x[child_j],
                        x_i.y - tree_center_of_mass_y[child_j],
                        x_i.z - tree_center_of_mass_z[child_j]
                    );
                    const double width_j = width / (2 << level);
                    norm_square = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
                    if (width_j / sqrt(norm_square) < opening_angle)
                    {
                        criteria_met = true;
                    }
                }

                // Traverse deeper
                if (!criteria_met)
                {
                    Stack *new_item = &(stack_pool[level + 1]);
                    new_item->node = child_j;
                    new_item->last = stack;
                    new_item->processed_children = -1;

                    stack = new_item;
                    level++;
                    break;
                }

                else
                {
                    const double R_norm = sqrt(
                        norm_square + softening_squared
                    );

                    const double temp_value = G / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x * tree_total_mass[child_j];
                    local_a.y -= temp_value * dr.y * tree_total_mass[child_j];
                    local_a.z -= temp_value * dr.z * tree_total_mass[child_j];

                    stack->processed_children = j;
                    continue;
                }
            }
        }

        if ((stack->processed_children + 1) >= tree_num_internal_children[stack->node])
        {
            Stack *parent = stack->last;
            if (!parent)
            {
                break;
            }            
            stack = parent;
            stack->processed_children += 1;
            level--;
        }
    }

    a[idx_i * 3 + 0] = local_a.x;
    a[idx_i * 3 + 1] = local_a.y;
    a[idx_i * 3 + 2] = local_a.z;

    return;
}

/**
 * \brief Compute the acceleration of the particles
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param actual_num_internal_nodes Number of internal nodes
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
IN_FILE int _compute_acceleration(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int actual_num_internal_nodes,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    int return_code;

    double *__restrict a_device = NULL;
    double *__restrict x_device = NULL;
    double *__restrict m_device = NULL;
    int64 *__restrict leaf_morton_indices_deepest_level_device = NULL;
    int *__restrict sorted_indices_device = NULL;
    int *__restrict tree_start_particle_sorted_idx_device = NULL;
    int *__restrict tree_num_particles_device = NULL;
    int *__restrict tree_num_internal_children_device = NULL;
    int *__restrict tree_idx_first_internal_child_device = NULL;
    double *__restrict tree_total_mass_device = NULL;
    double *__restrict tree_center_of_mass_x_device = NULL;
    double *__restrict tree_center_of_mass_y_device = NULL;
    double *__restrict tree_center_of_mass_z_device = NULL;
    hipError_t error;

    /* Allocate memory on GPU */
    error = hipMalloc((double **) &a_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &x_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &m_device, objects_count * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int64 **) &leaf_morton_indices_deepest_level_device, objects_count * sizeof(int64));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &sorted_indices_device, objects_count * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_start_particle_sorted_idx_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_particles_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_internal_children_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_idx_first_internal_child_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_total_mass_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_x_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_y_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_z_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMemcpy(x_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(m_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(leaf_morton_indices_deepest_level_device, leaf_morton_indices_deepest_level, objects_count * sizeof(int64), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(sorted_indices_device, sorted_indices, objects_count * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_start_particle_sorted_idx_device, tree_start_particle_sorted_idx, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_particles_device, tree_num_particles, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_internal_children_device, tree_num_internal_children, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_idx_first_internal_child_device, tree_idx_first_internal_child, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_total_mass_device, tree_total_mass, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_x_device, tree_center_of_mass_x, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_y_device, tree_center_of_mass_y, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_z_device, tree_center_of_mass_z, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    } 

    _acceleration_helper_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (
        a_device,
        objects_count,
        x_device,
        m_device,
        G,
        softening_length,
        opening_angle,
        width,
        leaf_morton_indices_deepest_level_device,
        sorted_indices_device,
        tree_start_particle_sorted_idx_device,
        tree_num_particles_device,
        tree_num_internal_children_device,
        tree_idx_first_internal_child_device,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    error = hipMemcpy(a, a_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_GPU_TO_CPU;
        goto err_memcpy_gpu_to_cpu;
    }

    hipFree(a_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);

    return SUCCESS;

err_memcpy_gpu_to_cpu:
err_memcpy_cpu_to_gpu:
err_gpu_memory:
    hipFree(a_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);
    return return_code;
}

extern "C"
{
    WIN32DLL_API int acceleration_barnes_hut_cuda(
        real *__restrict a,
        const System *__restrict system,
        AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;
    
        const int objects_count = system->objects_count;
        const real *__restrict x = system->x;
        const real *__restrict m = system->m;
        const real G = system->G;
        const real softening_length = acceleration_param->softening_length; 
        const real opening_angle = acceleration_param->opening_angle;
    
        /* Empty the input array */
        for (int i = 0; i < objects_count; i++)
        {
            a[i * 3 + 0] = 0.0;
            a[i * 3 + 1] = 0.0;
            a[i * 3 + 2] = 0.0;
        }
    
        /* Construct the octree */
        // Allocate memory
        real width;
        int64 *leaf_morton_indices_deepest_level;
        int *sorted_indices;
        int allocated_internal_nodes;
        int actual_num_internal_nodes;
        int *tree_start_particle_sorted_idx;
        int *tree_num_particles;
        int *tree_num_internal_children;
        int *tree_idx_first_internal_child;
        real *tree_total_mass;
        real *tree_center_of_mass_x;
        real *tree_center_of_mass_y;
        real *tree_center_of_mass_z;
    
        return_code = barnes_hut_setup_octree(
            &width,
            &allocated_internal_nodes,
            &actual_num_internal_nodes,
            objects_count,
            x,
            m,
            &leaf_morton_indices_deepest_level,
            &sorted_indices,
            &tree_start_particle_sorted_idx,
            &tree_num_particles,
            &tree_num_internal_children,
            &tree_idx_first_internal_child,
            &tree_total_mass,
            &tree_center_of_mass_x,
            &tree_center_of_mass_y,
            &tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_octree;
        }

        /* Compute the acceleration */ 
        return_code = _compute_acceleration(
            a,
            objects_count,
            x,
            m,
            G,
            softening_length,
            opening_angle,
            width,
            actual_num_internal_nodes,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            tree_start_particle_sorted_idx,
            tree_num_particles,
            tree_num_internal_children,
            tree_idx_first_internal_child,
            tree_total_mass,
            tree_center_of_mass_x,
            tree_center_of_mass_y,
            tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_acceleration;
        }

        /* Free the memory */
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return SUCCESS;

    err_acceleration:
    err_octree:
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return return_code;
    }
}

__global__ void memcpy_particles_array_double_to_float(
    const double *__restrict x_double,
    const double *__restrict m_double,
    const int objects_count,
    float *__restrict x,
    float *__restrict m
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    x[i * 3 + 0] = x_double[i * 3 + 0];
    x[i * 3 + 1] = x_double[i * 3 + 1];
    x[i * 3 + 2] = x_double[i * 3 + 2];
    m[i] = m_double[i];

    return;
}

__global__ void memcpy_nodes_array_double_to_float(
    const double *__restrict tree_total_mass_double,
    const double *__restrict tree_center_of_mass_x_double,
    const double *__restrict tree_center_of_mass_y_double,
    const double *__restrict tree_center_of_mass_z_double,
    const int actual_num_internal_nodes,
    float *__restrict tree_total_mass,
    float *__restrict tree_center_of_mass_x,
    float *__restrict tree_center_of_mass_y,
    float *__restrict tree_center_of_mass_z
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= actual_num_internal_nodes)
    {
        return;
    }

    tree_total_mass[i] = tree_total_mass_double[i];
    tree_center_of_mass_x[i] = tree_center_of_mass_x_double[i];
    tree_center_of_mass_y[i] = tree_center_of_mass_y_double[i];
    tree_center_of_mass_z[i] = tree_center_of_mass_z_double[i];

    return;
}

/**
 * \brief Helper kernel function for computing the acceleration of one particle in single precision
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
 __global__ void _acceleration_helper_float_kernel(
    double *__restrict a,
    const int objects_count,
    const float *__restrict x,
    const float *__restrict m,
    const float G,
    const float softening_length,
    const float opening_angle,
    const float width,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const float *__restrict tree_total_mass,
    const float *__restrict tree_center_of_mass_x,
    const float *__restrict tree_center_of_mass_y,
    const float *__restrict tree_center_of_mass_z
)
{
    typedef struct Stack
    {
        int node;
        int processed_children;
        struct Stack *last;
    } Stack;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    const int idx_i = sorted_indices[i];
    const int64 morton_index_i = leaf_morton_indices_deepest_level[idx_i];
    const float3 x_i = make_float3(x[idx_i * 3 + 0], x[idx_i * 3 + 1], x[idx_i * 3 + 2]);
    const float softening_squared = softening_length * softening_length;

    Stack stack_pool[MORTON_MAX_LEVEL];
    Stack *stack = &(stack_pool[0]);
    stack->processed_children = -1;
    stack->last = NULL;
    stack->node = 0;
    float3 local_a = make_float3(0.0, 0.0, 0.0);

    int level = 1;

    /* Tree walk */
    while (true)
    {
        const int node = stack->node;
        for (int j = (stack->processed_children) + 1; j < tree_num_internal_children[node]; j++)
        {
            const int child_j = tree_idx_first_internal_child[node] + j;
            const int num_children_j = tree_num_internal_children[child_j];
            const int start_idx_j = tree_start_particle_sorted_idx[child_j];

            /* Leaf node */
            if (num_children_j <= 0)
            {
                const int num_particles_j = tree_num_particles[child_j];
                for (int k = 0; k < num_particles_j; k++)
                {
                    const int idx_j = sorted_indices[start_idx_j + k];
                    if (idx_i == idx_j)
                    {
                        continue;
                    }

                    // Calculate \vec{R} and its norm
                    const float3 dr = make_float3(
                        x_i.x - x[idx_j * 3 + 0],
                        x_i.y - x[idx_j * 3 + 1],
                        x_i.z - x[idx_j * 3 + 2]
                    );
                    const float R_norm = sqrt(
                        dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + softening_squared
                    );

                    // Calculate the acceleration
                    const float temp_value = G * m[idx_j] / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x;
                    local_a.y -= temp_value * dr.y;
                    local_a.z -= temp_value * dr.z;
                }

                stack->processed_children = j;
                continue;
            }

            /* Internal node */
            else
            {
                bool criteria_met = false;

                // If object i is included, then we need to traverse deeper
                const bool is_included = _check_if_included(
                    morton_index_i,
                    leaf_morton_indices_deepest_level[sorted_indices[start_idx_j]],
                    level
                );

                // Check Barnes-Hut criteria
                float3 dr;
                float norm_square;
                if (!is_included)
                {
                    dr = make_float3(
                        x_i.x - tree_center_of_mass_x[child_j],
                        x_i.y - tree_center_of_mass_y[child_j],
                        x_i.z - tree_center_of_mass_z[child_j]
                    );
                    const float width_j = width / (2 << level);
                    norm_square = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
                    if (width_j / sqrt(norm_square) < opening_angle)
                    {
                        criteria_met = true;
                    }
                }

                // Traverse deeper
                if (!criteria_met)
                {
                    Stack *new_item = &(stack_pool[level + 1]);
                    new_item->node = child_j;
                    new_item->last = stack;
                    new_item->processed_children = -1;

                    stack = new_item;
                    level++;
                    break;
                }

                else
                {
                    const float R_norm = sqrt(
                        norm_square + softening_squared
                    );

                    const float temp_value = G / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x * tree_total_mass[child_j];
                    local_a.y -= temp_value * dr.y * tree_total_mass[child_j];
                    local_a.z -= temp_value * dr.z * tree_total_mass[child_j];

                    stack->processed_children = j;
                    continue;
                }
            }
        }

        if ((stack->processed_children + 1) >= tree_num_internal_children[stack->node])
        {
            Stack *parent = stack->last;
            if (!parent)
            {
                break;
            }            
            stack = parent;
            stack->processed_children += 1;
            level--;
        }
    }

    a[idx_i * 3 + 0] = local_a.x;
    a[idx_i * 3 + 1] = local_a.y;
    a[idx_i * 3 + 2] = local_a.z;

    return;
}

/**
 * \brief Compute the acceleration of the particles with single precision
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param actual_num_internal_nodes Number of internal nodes
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
 IN_FILE int _compute_acceleration_float(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int actual_num_internal_nodes,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    int return_code;

    double *__restrict a_double_device = NULL;
    double *__restrict x_double_device = NULL;
    double *__restrict m_double_device = NULL;
    double *__restrict tree_total_mass_double_device = NULL;
    double *__restrict tree_center_of_mass_x_double_device = NULL;
    double *__restrict tree_center_of_mass_y_double_device = NULL;
    double *__restrict tree_center_of_mass_z_double_device = NULL;

    float *__restrict x_device = NULL;
    float *__restrict m_device = NULL;
    int64 *__restrict leaf_morton_indices_deepest_level_device = NULL;
    int *__restrict sorted_indices_device = NULL;
    int *__restrict tree_start_particle_sorted_idx_device = NULL;
    int *__restrict tree_num_particles_device = NULL;
    int *__restrict tree_num_internal_children_device = NULL;
    int *__restrict tree_idx_first_internal_child_device = NULL;
    float *__restrict tree_total_mass_device = NULL;
    float *__restrict tree_center_of_mass_x_device = NULL;
    float *__restrict tree_center_of_mass_y_device = NULL;
    float *__restrict tree_center_of_mass_z_device = NULL;
    hipError_t error;

    /* Allocate memory on GPU */
    error = hipMalloc((double **) &a_double_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &x_double_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &m_double_device, objects_count * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_total_mass_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_x_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_y_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_z_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMalloc((float **) &x_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &m_device, objects_count * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int64 **) &leaf_morton_indices_deepest_level_device, objects_count * sizeof(int64));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &sorted_indices_device, objects_count * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_start_particle_sorted_idx_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_particles_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_internal_children_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_idx_first_internal_child_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_total_mass_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_x_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_y_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_z_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMemcpy(x_double_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(m_double_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_total_mass_double_device, tree_total_mass, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_x_double_device, tree_center_of_mass_x, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_y_double_device, tree_center_of_mass_y, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_z_double_device, tree_center_of_mass_z, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    } 

    error = hipMemcpy(leaf_morton_indices_deepest_level_device, leaf_morton_indices_deepest_level, objects_count * sizeof(int64), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(sorted_indices_device, sorted_indices, objects_count * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_start_particle_sorted_idx_device, tree_start_particle_sorted_idx, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_particles_device, tree_num_particles, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_internal_children_device, tree_num_internal_children, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_idx_first_internal_child_device, tree_idx_first_internal_child, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }

    memcpy_particles_array_double_to_float <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        x_double_device,
        m_double_device,
        objects_count,
        x_device,
        m_device
    );

    memcpy_nodes_array_double_to_float <<< (actual_num_internal_nodes + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        tree_total_mass_double_device,
        tree_center_of_mass_x_double_device,
        tree_center_of_mass_y_double_device,
        tree_center_of_mass_z_double_device,
        actual_num_internal_nodes,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    hipFree(x_double_device);
    hipFree(m_double_device);
    hipFree(tree_total_mass_double_device);
    hipFree(tree_center_of_mass_x_double_device);
    hipFree(tree_center_of_mass_y_double_device);
    hipFree(tree_center_of_mass_z_double_device);

    _acceleration_helper_float_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (
        a_double_device,
        objects_count,
        x_device,
        m_device,
        G,
        softening_length,
        opening_angle,
        width,
        leaf_morton_indices_deepest_level_device,
        sorted_indices_device,
        tree_start_particle_sorted_idx_device,
        tree_num_particles_device,
        tree_num_internal_children_device,
        tree_idx_first_internal_child_device,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    error = hipMemcpy(a, a_double_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_GPU_TO_CPU;
        goto err_memcpy_gpu_to_cpu;
    }

    hipFree(a_double_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);

    return SUCCESS;

err_memcpy_cpu_to_gpu:
err_gpu_memory:
    hipFree(x_double_device);
    hipFree(m_double_device);
    hipFree(tree_total_mass_double_device);
    hipFree(tree_center_of_mass_x_double_device);
    hipFree(tree_center_of_mass_y_double_device);
    hipFree(tree_center_of_mass_z_double_device);
err_memcpy_gpu_to_cpu:
    hipFree(a_double_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);
    return return_code;
}


extern "C"
{
    WIN32DLL_API int acceleration_barnes_hut_cuda_float(
        real *__restrict a,
        const System *__restrict system,
        AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;
    
        const int objects_count = system->objects_count;
        const real *__restrict x = system->x;
        const real *__restrict m = system->m;
        const real G = system->G;
        const real softening_length = acceleration_param->softening_length; 
        const real opening_angle = acceleration_param->opening_angle;
    
        /* Empty the input array */
        for (int i = 0; i < objects_count; i++)
        {
            a[i * 3 + 0] = 0.0;
            a[i * 3 + 1] = 0.0;
            a[i * 3 + 2] = 0.0;
        }
    
        /* Construct the octree */
        // Allocate memory
        real width;
        int64 *leaf_morton_indices_deepest_level;
        int *sorted_indices;
        int allocated_internal_nodes;
        int actual_num_internal_nodes;
        int *tree_start_particle_sorted_idx;
        int *tree_num_particles;
        int *tree_num_internal_children;
        int *tree_idx_first_internal_child;
        real *tree_total_mass;
        real *tree_center_of_mass_x;
        real *tree_center_of_mass_y;
        real *tree_center_of_mass_z;
    
        return_code = barnes_hut_setup_octree(
            &width,
            &allocated_internal_nodes,
            &actual_num_internal_nodes,
            objects_count,
            x,
            m,
            &leaf_morton_indices_deepest_level,
            &sorted_indices,
            &tree_start_particle_sorted_idx,
            &tree_num_particles,
            &tree_num_internal_children,
            &tree_idx_first_internal_child,
            &tree_total_mass,
            &tree_center_of_mass_x,
            &tree_center_of_mass_y,
            &tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_octree;
        }

        /* Compute the acceleration */ 
        return_code = _compute_acceleration_float(
            a,
            objects_count,
            x,
            m,
            G,
            softening_length,
            opening_angle,
            width,
            actual_num_internal_nodes,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            tree_start_particle_sorted_idx,
            tree_num_particles,
            tree_num_internal_children,
            tree_idx_first_internal_child,
            tree_total_mass,
            tree_center_of_mass_x,
            tree_center_of_mass_y,
            tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_acceleration;
        }

        /* Free the memory */
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return SUCCESS;

    err_acceleration:
    err_octree:
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return return_code;
    }
}
