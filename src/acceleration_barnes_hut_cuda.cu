#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "acceleration_cuda.cuh"
#include "error.h"
#include "gravity_sim.h"
#include "math_functions.h"

#define MAX_NUM_PARTICLES_PER_LEAF 1 // Note: Potential optimization, fix this to 1 we can remove some loops and we may not need tree_num_particles array
#define MORTON_MAX_LEVEL 21 // Maximum level for 64-bit Morton index, don't change

/**
 * \brief Calculate the bounding box of the system
 * 
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param center 3D vector of the center of the bounding box
 * \param width Width of the bounding box
 */
IN_FILE void _calculate_bounding_box(
    const int objects_count,
    const double *__restrict x,
    double *__restrict center,
    double *__restrict width
)
{
    /* Find the width of the bounding box */
    double min_x = x[0];
    double max_x = x[0];
    double min_y = x[1];
    double max_y = x[1];
    double min_z = x[2];
    double max_z = x[2];

    for (int i = 1; i < objects_count; i++)
    {
        min_x = fmin(min_x, x[i * 3 + 0]);
        max_x = fmax(max_x, x[i * 3 + 0]);
        min_y = fmin(min_y, x[i * 3 + 1]);
        max_y = fmax(max_y, x[i * 3 + 1]);
        min_z = fmin(min_z, x[i * 3 + 2]);
        max_z = fmax(max_z, x[i * 3 + 2]);
    }

    center[0] = (max_x + min_x) / 2.0;
    center[1] = (max_y + min_y) / 2.0;
    center[2] = (max_z + min_z) / 2.0;

    double width_x = max_x - min_x;
    double width_y = max_y - min_y;
    double width_z = max_z - min_z;
    *width = fmax(fmax(width_x, width_y), width_z);
}

/**
 * \brief Compute the 3D Morton indices at level 21 using magic number
 * 
 * \param morton_indices Array of Morton indices
 * \param object_count Number of objects
 * \param x Array of position vectors
 * \param center 3D vector of the center of the bounding box
 * \param width Width of the bounding box
 * 
 * \ref https://stackoverflow.com/a/18528775, Stack Overflow
 */
IN_FILE void _compute_3d_morton_indices_level_21(
    int64 *__restrict morton_indices,
    const int object_count,
    const double *__restrict x,
    const double *__restrict center,
    const double width
)
{
    for (int i = 0; i < object_count; i++)
    {
        /* Normalize the position */
        const double x_i = (x[i * 3 + 0] - center[0]) / width + 0.5;
        const double y_i = (x[i * 3 + 1] - center[1]) / width + 0.5;
        const double z_i = (x[i * 3 + 2] - center[2]) / width + 0.5;

        /* Compute the morton indices */
        int64 n_x = x_i * (1 << 21);
        int64 n_y = y_i * (1 << 21);
        int64 n_z = z_i * (1 << 21);

        n_x &= 0x1fffff;
        n_x = (n_x | n_x << 32) & 0x1f00000000ffff;
        n_x = (n_x | n_x << 16) & 0x1f0000ff0000ff;
        n_x = (n_x | n_x << 8)  & 0x100f00f00f00f00f;
        n_x = (n_x | n_x << 4)  & 0x10c30c30c30c30c3;
        n_x = (n_x | n_x << 2)  & 0x1249249249249249;
        
        n_y &= 0x1fffff;
        n_y = (n_y | n_y << 32) & 0x1f00000000ffff;
        n_y = (n_y | n_y << 16) & 0x1f0000ff0000ff;
        n_y = (n_y | n_y << 8)  & 0x100f00f00f00f00f;
        n_y = (n_y | n_y << 4)  & 0x10c30c30c30c30c3;
        n_y = (n_y | n_y << 2)  & 0x1249249249249249;

        n_z &= 0x1fffff;
        n_z = (n_z | n_z << 32) & 0x1f00000000ffff;
        n_z = (n_z | n_z << 16) & 0x1f0000ff0000ff;
        n_z = (n_z | n_z << 8)  & 0x100f00f00f00f00f;
        n_z = (n_z | n_z << 4)  & 0x10c30c30c30c30c3;
        n_z = (n_z | n_z << 2)  & 0x1249249249249249;

        morton_indices[i] = n_x | (n_y << 1) | (n_z << 2);
    }
}

/**
 * \brief Perform radix sort on the particles based on their Morton indices
 * 
 * \param object_count Number of objects
 * \param morton_indices Array of Morton indices
 * \param indices Array of indices
 * \param level Level of the Morton indices
 * 
 * \retval SUCCESS if successful
 * \retval ERROR_BARNES_HUT_RADIX_SORT_MEMORY_ALLOC if memory allocation fails
 */
IN_FILE int _radix_sort_particles_morton_index(
    const int object_count,
    int64 *__restrict morton_indices,
    int *__restrict indices,
    const int level
)
{
    int return_code;

    /* Calculate constnats */
    const int RADIX_BITS = 9;
    const int RADIX_SIZE = 1 << RADIX_BITS;
    const int RADIX_MASK = RADIX_SIZE - 1;
    
    const int num_significant_bits = 3 * level;
    const int num_passes = (num_significant_bits + RADIX_BITS - 1) / RADIX_BITS;

    // Flag to indicate whether the sorted array is in temp arrays
    // This can reduce the number of memcpy to O(1) instead of O(num_passes)
    bool is_temp = false; 

    /* Allocate memory */
    int64 *__restrict temp_morton_indices = (int64*) malloc(object_count * sizeof(int64));
    int *__restrict temp_indices = (int*) malloc(object_count * sizeof(int));
    int *__restrict count = (int*) malloc(RADIX_SIZE * sizeof(int));
    if (!temp_morton_indices || !temp_indices || !count)
    {
        return_code = ERROR_BARNES_HUT_RADIX_SORT_MEMORY_ALLOC;
        goto err_memory;
    }
    
    /* Perform LSB radix sort */    
    for (int i = 0; i < num_passes; i++) 
    {
        // Empty count array
        for (int j = 0; j < RADIX_SIZE; j++)
        {
            count[j] = 0;
        }

        // Calculate shift for this pass (start from least significant bits)
        const int shift = i * RADIX_BITS;
        
        // Count occurrences of each radix value
        if (is_temp)
        {
            for (int j = 0; j < object_count; j++) 
            {
                count[(temp_morton_indices[j] >> shift) & RADIX_MASK]++;
            }
        }
        else
        {
            for (int j = 0; j < object_count; j++) 
            {
                count[(morton_indices[j] >> shift) & RADIX_MASK]++;
            }
        }

        // Get cumulative count
        int total = 0;
        for (int j = 0; j < RADIX_SIZE; j++) 
        {
            int old_count = count[j];
            count[j] = total;
            total += old_count;
        }
        
        // Sort elements into temporary arrays
        if (is_temp)
        {
            for (int j = 0; j < object_count; j++) 
            {
                const int dest = count[(temp_morton_indices[j] >> shift) & RADIX_MASK]++;
                
                morton_indices[dest] = temp_morton_indices[j];
                indices[dest] = temp_indices[j];
            }
        }
        else
        {
            for (int j = 0; j < object_count; j++) 
            {
                const int dest = count[(morton_indices[j] >> shift) & RADIX_MASK]++;
                
                temp_morton_indices[dest] = morton_indices[j];
                temp_indices[dest] = indices[j];
            }
        }
        
        is_temp = !is_temp;
    }

    // Copy the sorted array to the original array
    if (is_temp)
    {
        memcpy(morton_indices, temp_morton_indices, object_count * sizeof(int64));
        memcpy(indices, temp_indices, object_count * sizeof(int));
    }
    
    free(count);
    free(temp_morton_indices);
    free(temp_indices);

    return SUCCESS;

err_memory:
    free(count);
    free(temp_morton_indices);
    free(temp_indices);

    return return_code;
}

/**
 * \brief Perform binary search to find the number of particles in each octant
 * 
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param node_morton_index_level Morton index of the node
 * \param start_idx Start index of the particles in the node
 * \param end_idx End index of the particles in the node
 * \param leaf_level Level of the leaf nodes
 * \param num_particles_per_octant Array to store the number of particles in each octant
 */
IN_FILE void _binary_search_num_particles_per_octant(
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int64 node_morton_index_level,
    const int start_idx,
    const int end_idx,
    const int leaf_level,
    int *__restrict num_particles_per_octant
)
{
    const int64 prefix = node_morton_index_level * 8;
    const int level_shift = 3 * (MORTON_MAX_LEVEL - leaf_level);

    int cumulative_count = 0;

    for (int i = 0; i < 8; i++)
    {
        // Binary search for the index of last i
        int left = start_idx + cumulative_count;
        int right = end_idx;
        while (left <= right)
        {
            const int mid = left + (right - left) / 2;
            const int mid_octant = ((leaf_morton_indices_deepest_level[mid] >> level_shift) - prefix);

            if (mid_octant < 0 || mid_octant > 7)
            {
                printf("Warning: mid_octant out of range: %d\n", mid_octant);
            }

            if (mid_octant == i && (mid == end_idx || (((leaf_morton_indices_deepest_level[mid + 1] >> level_shift) - prefix)) > i))
            {
                num_particles_per_octant[i] = mid - (start_idx + cumulative_count) + 1;
                cumulative_count += num_particles_per_octant[i];
                break;
            }
            else if (mid_octant <= i)
            {
                left = mid + 1;
            }
            else
            {
                right = mid - 1;
            }
        }
    }
}

/**
 * \brief Set up a new internal node
 * 
 * \param allocated_internal_nodes Pointer to the number of allocated internal nodes
 * \param internal_node_count Pointer to the number of internal nodes
 * \param level Node level
 * \param width Node width
 * \param node Node index
 * \param node_morton_index_level Morton index of the node at the current level
 * \param leaf_morton_indices_deepest_level Pointer to array of Morton indices
 * \param tree_num_internal_children Pointer to array of number of internal children
 * \param tree_idx_first_internal_child Pointer to array of index of the first internal child
 * \param tree_start_particle_sorted_idx Pointer to array of start index of particles in the node
 * \param tree_num_particles Pointer to array of number of particles in the node
 * \param tree_total_mass Pointer to array of total mass of the node
 * \param tree_center_of_mass_x Pointer to array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Pointer to array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Pointer to array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 * \retval error_code if there is an error
 */
IN_FILE int _setup_node(
    int *__restrict allocated_internal_nodes,
    int *__restrict internal_node_count,
    const int level,
    const double width,
    const int node,
    const int64 node_morton_index_level,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    int **tree_num_internal_children,
    int **tree_idx_first_internal_child,
    int **tree_start_particle_sorted_idx,
    int **tree_num_particles,
    double **tree_total_mass,
    double **tree_center_of_mass_x,
    double **tree_center_of_mass_y,
    double **tree_center_of_mass_z
)
{
    int return_code;

    int num_particles_per_octant[8] = {0};
    const int start_idx = (*tree_start_particle_sorted_idx)[node];
    const int end_idx = start_idx + (*tree_num_particles)[node] - 1;
    const int child_level = level + 1;
    _binary_search_num_particles_per_octant(
        leaf_morton_indices_deepest_level,
        node_morton_index_level,
        start_idx,
        end_idx,
        child_level,
        num_particles_per_octant
    );

    bool first_child_found = false;
    int cumulative_count = 0;
    for (int i = 0; i < 8; i++)
    {
        if (num_particles_per_octant[i] == 0)
        {
            continue;
        }

        const int child = *internal_node_count;

        // Reallocate memory if necessary
        if (child >= *allocated_internal_nodes)
        {
            *allocated_internal_nodes *= 2;
            int *tmp_tree_num_internal_children = (int*) realloc(*tree_num_internal_children, *allocated_internal_nodes * sizeof(int));
            int *tmp_tree_idx_first_internal_child = (int*) realloc(*tree_idx_first_internal_child, *allocated_internal_nodes * sizeof(int));
            int *tmp_tree_start_particle_sorted_idx = (int*) realloc(*tree_start_particle_sorted_idx, *allocated_internal_nodes * sizeof(int));
            int *tmp_tree_num_particles = (int*) realloc(*tree_num_particles, *allocated_internal_nodes * sizeof(int));
            double *tmp_tree_total_mass = (double*) realloc(*tree_total_mass, *allocated_internal_nodes * sizeof(double));
            double *tmp_tree_center_of_mass_x = (double*) realloc(*tree_center_of_mass_x, *allocated_internal_nodes * sizeof(double));
            double *tmp_tree_center_of_mass_y = (double*) realloc(*tree_center_of_mass_y, *allocated_internal_nodes * sizeof(double));
            double *tmp_tree_center_of_mass_z = (double*) realloc(*tree_center_of_mass_z, *allocated_internal_nodes * sizeof(double));

            if (
                !tmp_tree_num_internal_children ||
                !tmp_tree_idx_first_internal_child ||
                !tmp_tree_start_particle_sorted_idx ||
                !tmp_tree_num_particles ||
                !tmp_tree_total_mass ||
                !tmp_tree_center_of_mass_x ||
                !tmp_tree_center_of_mass_y ||
                !tmp_tree_center_of_mass_z
            )
            {
                return_code = ERROR_BARNES_HUT_SETUP_NODE_MEMORY_REALLOC;
                goto err_memory_realloc;
            }

            *tree_num_internal_children = tmp_tree_num_internal_children;
            *tree_idx_first_internal_child = tmp_tree_idx_first_internal_child;
            *tree_start_particle_sorted_idx = tmp_tree_start_particle_sorted_idx;
            *tree_num_particles = tmp_tree_num_particles;
            *tree_total_mass = tmp_tree_total_mass;
            *tree_center_of_mass_x = tmp_tree_center_of_mass_x;
            *tree_center_of_mass_y = tmp_tree_center_of_mass_y;
            *tree_center_of_mass_z = tmp_tree_center_of_mass_z;
        }

        if (!first_child_found)
        {
            first_child_found = true;
            (*tree_idx_first_internal_child)[node] = child;
            (*tree_num_internal_children)[node] = 0;
        }

        // Create a new internal node
        (*tree_num_internal_children)[node] += 1;
        (*tree_start_particle_sorted_idx)[child] = start_idx + cumulative_count;
        (*tree_num_particles)[child] = num_particles_per_octant[i];

        (*tree_center_of_mass_x)[child] = (*tree_center_of_mass_x)[node];
        (*tree_center_of_mass_y)[child] = (*tree_center_of_mass_y)[node];
        (*tree_center_of_mass_z)[child] = (*tree_center_of_mass_z)[node];

        const double child_half_width = width / (2 << child_level);

        if (i & 1)
        {
            (*tree_center_of_mass_x)[child] += child_half_width;
        }
        else
        {
            (*tree_center_of_mass_x)[child] -= child_half_width;
        }

        if (i & 2)
        {
            (*tree_center_of_mass_y)[child] += child_half_width;
        }
        else
        {
            (*tree_center_of_mass_y)[child] -= child_half_width;
        }

        if (i & 4)
        {
            (*tree_center_of_mass_z)[child] += child_half_width;
        }
        else
        {
            (*tree_center_of_mass_z)[child] -= child_half_width;
        }
        
        // Update counters
        (*internal_node_count) += 1;
        cumulative_count += num_particles_per_octant[i];
    }

    return SUCCESS;

err_memory_realloc:
    return return_code;
}

/**
 * \brief Construct the octree
 * 
 * \param allocated_internal_nodes Pointer to the number of allocated internal nodes
 * \param actual_num_internal_nodes Pointer to the count of internal nodes
 * \param x Array of position vectors
 * \param m Array of masses
 * \param width Width of the bounding box
 * \param sorted_indices Array of sorted indices
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param morton_max_level Maximum level of the Morton indices
 * \param tree_start_particle_sorted_idx Pointer to array of start index of particles in the node
 * \param tree_num_particles Pointer to array of number of particles in the node
 * \param tree_num_internal_children Pointer to array of number of internal children
 * \param tree_idx_first_internal_child Pointer to array of index of the first internal child
 * \param tree_total_mass Pointer to array of total mass of the node
 * \param tree_center_of_mass_x Pointer to array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Pointer to array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Pointer to array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 * \retval error_code if there is an error
 */
IN_FILE int _construct_octree(
    int *__restrict allocated_internal_nodes,
    int *__restrict actual_num_internal_nodes,
    const double *__restrict x,
    const double *__restrict m,
    const double width,
    const int *__restrict sorted_indices,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int morton_max_level,
    int **tree_start_particle_sorted_idx,
    int **tree_num_particles,
    int **tree_num_internal_children,
    int **tree_idx_first_internal_child,
    double **tree_total_mass,
    double **tree_center_of_mass_x,
    double **tree_center_of_mass_y,
    double **tree_center_of_mass_z
)
{
    typedef struct Stack
    {
        int node;
        int processed_children;
        struct Stack *last;
        double total_mass;
        double mass_times_distance[3];
    } Stack;

    int return_code;

    /* Create a stack pool */
    Stack stack_pool[MORTON_MAX_LEVEL];
    Stack *stack = &(stack_pool[0]);

    stack->node = 0;
    stack->processed_children = -1;
    stack->last = NULL;
    stack->total_mass = 0.0;
    stack->mass_times_distance[0] = 0.0;
    stack->mass_times_distance[1] = 0.0;
    stack->mass_times_distance[2] = 0.0;

    int level = 0;
    int internal_node_count = 1;

    /* Set up the root node */
    return_code = _setup_node(
        allocated_internal_nodes,
        &internal_node_count,
        level,
        width,
        stack->node,
        0,
        leaf_morton_indices_deepest_level,
        tree_num_internal_children,
        tree_idx_first_internal_child,
        tree_start_particle_sorted_idx,
        tree_num_particles,
        tree_total_mass,
        tree_center_of_mass_x,
        tree_center_of_mass_y,
        tree_center_of_mass_z
    );
    if (return_code != SUCCESS)
    {
        goto err_setup_node;
    }

    level++;

    while (true)
    {
        const int node = stack->node;
        for (int i = stack->processed_children + 1; i < (*tree_num_internal_children)[node]; i++)
        {
            const int child = (*tree_idx_first_internal_child)[node] + i;
            const int start_idx = (*tree_start_particle_sorted_idx)[child];
            const int num_particles = (*tree_num_particles)[child];

            const int64 child_morton_index_level = (leaf_morton_indices_deepest_level[start_idx] >> (3 * (MORTON_MAX_LEVEL - level)));

            if (num_particles <= MAX_NUM_PARTICLES_PER_LEAF || level >= morton_max_level)
            {
                // Leaf node
                (*tree_num_internal_children)[child] = 0;

                // Update the stack
                stack->processed_children = i;

                for (int j = 0; j < num_particles; j++)
                {
                    const int particle_idx = sorted_indices[start_idx + j];
                    stack->total_mass += m[particle_idx];
                    stack->mass_times_distance[0] += m[particle_idx] * x[particle_idx * 3 + 0];
                    stack->mass_times_distance[1] += m[particle_idx] * x[particle_idx * 3 + 1];
                    stack->mass_times_distance[2] += m[particle_idx] * x[particle_idx * 3 + 2];
                }

                continue;
            }
            else
            {
                return_code = _setup_node(
                    allocated_internal_nodes,
                    &internal_node_count,
                    level,
                    width,
                    child,
                    child_morton_index_level,
                    leaf_morton_indices_deepest_level,
                    tree_num_internal_children,
                    tree_idx_first_internal_child,
                    tree_start_particle_sorted_idx,
                    tree_num_particles,
                    tree_total_mass,
                    tree_center_of_mass_x,
                    tree_center_of_mass_y,
                    tree_center_of_mass_z
                );
                if (return_code != SUCCESS)
                {
                    goto err_setup_node;
                }

                Stack *new_item = &(stack_pool[level + 1]);
                new_item->node = child;
                new_item->last = stack;
                new_item->processed_children = -1;
                new_item->total_mass = 0.0;
                new_item->mass_times_distance[0] = 0.0;
                new_item->mass_times_distance[1] = 0.0;
                new_item->mass_times_distance[2] = 0.0;

                stack = new_item;
                level++;

                break;
            }
        }

        if ((stack->processed_children + 1) >= (*tree_num_internal_children)[stack->node])
        {
            /* Update center of mass */
            (*tree_total_mass)[node] = stack->total_mass;
            (*tree_center_of_mass_x)[node] = stack->mass_times_distance[0] / stack->total_mass;
            (*tree_center_of_mass_y)[node] = stack->mass_times_distance[1] / stack->total_mass;
            (*tree_center_of_mass_z)[node] = stack->mass_times_distance[2] / stack->total_mass;

            Stack *parent = stack->last;
            if (!parent)
            {
                break;
            }

            parent->total_mass += stack->total_mass;
            parent->mass_times_distance[0] += stack->mass_times_distance[0];
            parent->mass_times_distance[1] += stack->mass_times_distance[1];
            parent->mass_times_distance[2] += stack->mass_times_distance[2];
            
            stack = parent;
            stack->processed_children += 1;
            level--;
        }
    }

    *actual_num_internal_nodes = internal_node_count;

    return SUCCESS;

err_setup_node:
    return return_code;
}

/**
 * \brief Check if two Morton indices are included in the same octant
 * 
 * \param morton_index_i Morton index of the first object at the deepest level
 * \param morton_index_j Morton index of the second object at the deepest level
 * \param level Level of the Morton indices
 */
 inline __device__ bool _check_if_included(
    const int64 morton_index_i,
    const int64 morton_index_j,
    const int level
)
{
    return (morton_index_i >> (3 * (MORTON_MAX_LEVEL - level))) == (morton_index_j >> (3 * (MORTON_MAX_LEVEL - level)));
}

/**
 * \brief Helper kernel function for computing the acceleration of one particle
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
__global__ void _acceleration_helper_kernel(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    typedef struct Stack
    {
        int node;
        int processed_children;
        struct Stack *last;
    } Stack;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    const int idx_i = sorted_indices[i];
    const int64 morton_index_i = leaf_morton_indices_deepest_level[idx_i];
    const double3 x_i = make_double3(x[idx_i * 3 + 0], x[idx_i * 3 + 1], x[idx_i * 3 + 2]);
    const double softening_squared = softening_length * softening_length;
    
    Stack stack_pool[MORTON_MAX_LEVEL];
    Stack *stack = &(stack_pool[0]);
    stack->processed_children = -1;
    stack->last = NULL;
    stack->node = 0;
    double3 local_a = make_double3(0.0, 0.0, 0.0);

    int level = 1;

    /* Tree walk */
    while (true)
    {
        const int node = stack->node;
        for (int j = (stack->processed_children) + 1; j < tree_num_internal_children[node]; j++)
        {
            const int child_j = tree_idx_first_internal_child[node] + j;
            const int num_children_j = tree_num_internal_children[child_j];
            const int start_idx_j = tree_start_particle_sorted_idx[child_j];

            /* Leaf node */
            if (num_children_j <= 0)
            {
                const int num_particles_j = tree_num_particles[child_j];
                for (int k = 0; k < num_particles_j; k++)
                {
                    const int idx_j = sorted_indices[start_idx_j + k];
                    if (idx_i == idx_j)
                    {
                        continue;
                    }

                    // Calculate \vec{R} and its norm
                    const double3 dr = make_double3(
                        x_i.x - x[idx_j * 3 + 0],
                        x_i.y - x[idx_j * 3 + 1],
                        x_i.z - x[idx_j * 3 + 2]
                    );
                    const double R_norm = sqrt(
                        dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + softening_squared
                    );

                    // Calculate the acceleration
                    const double temp_value = G * m[idx_j] / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x;
                    local_a.y -= temp_value * dr.y;
                    local_a.z -= temp_value * dr.z;
                }

                stack->processed_children = j;
                continue;
            }

            /* Internal node */
            else
            {
                bool criteria_met = false;

                // If object i is included, then we need to traverse deeper
                const bool is_included = _check_if_included(
                    morton_index_i,
                    leaf_morton_indices_deepest_level[sorted_indices[start_idx_j]],
                    level
                );

                // Check Barnes-Hut criteria
                double3 dr;
                double norm_square;
                if (!is_included)
                {
                    dr = make_double3(
                        x_i.x - tree_center_of_mass_x[child_j],
                        x_i.y - tree_center_of_mass_y[child_j],
                        x_i.z - tree_center_of_mass_z[child_j]
                    );
                    const double width_j = width / (2 << level);
                    norm_square = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
                    if (width_j / sqrt(norm_square) < opening_angle)
                    {
                        criteria_met = true;
                    }
                }

                // Traverse deeper
                if (!criteria_met)
                {
                    Stack *new_item = &(stack_pool[level + 1]);
                    new_item->node = child_j;
                    new_item->last = stack;
                    new_item->processed_children = -1;

                    stack = new_item;
                    level++;
                    break;
                }

                else
                {
                    const double R_norm = sqrt(
                        norm_square + softening_squared
                    );

                    const double temp_value = G / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x * tree_total_mass[child_j];
                    local_a.y -= temp_value * dr.y * tree_total_mass[child_j];
                    local_a.z -= temp_value * dr.z * tree_total_mass[child_j];

                    stack->processed_children = j;
                    continue;
                }
            }
        }

        if ((stack->processed_children + 1) >= tree_num_internal_children[stack->node])
        {
            Stack *parent = stack->last;
            if (!parent)
            {
                break;
            }            
            stack = parent;
            stack->processed_children += 1;
            level--;
        }
    }

    a[idx_i * 3 + 0] = local_a.x;
    a[idx_i * 3 + 1] = local_a.y;
    a[idx_i * 3 + 2] = local_a.z;

    return;
}

/**
 * \brief Compute the acceleration of the particles
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param actual_num_internal_nodes Number of internal nodes
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
IN_FILE int _compute_acceleration(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int actual_num_internal_nodes,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    int return_code;

    double *__restrict a_device = NULL;
    double *__restrict x_device = NULL;
    double *__restrict m_device = NULL;
    int64 *__restrict leaf_morton_indices_deepest_level_device = NULL;
    int *__restrict sorted_indices_device = NULL;
    int *__restrict tree_start_particle_sorted_idx_device = NULL;
    int *__restrict tree_num_particles_device = NULL;
    int *__restrict tree_num_internal_children_device = NULL;
    int *__restrict tree_idx_first_internal_child_device = NULL;
    double *__restrict tree_total_mass_device = NULL;
    double *__restrict tree_center_of_mass_x_device = NULL;
    double *__restrict tree_center_of_mass_y_device = NULL;
    double *__restrict tree_center_of_mass_z_device = NULL;
    hipError_t error;

    /* Allocate memory on GPU */
    error = hipMalloc((double **) &a_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &x_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &m_device, objects_count * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int64 **) &leaf_morton_indices_deepest_level_device, objects_count * sizeof(int64));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &sorted_indices_device, objects_count * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_start_particle_sorted_idx_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_particles_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_internal_children_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_idx_first_internal_child_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_total_mass_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_x_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_y_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_z_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMemcpy(x_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(m_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(leaf_morton_indices_deepest_level_device, leaf_morton_indices_deepest_level, objects_count * sizeof(int64), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(sorted_indices_device, sorted_indices, objects_count * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_start_particle_sorted_idx_device, tree_start_particle_sorted_idx, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_particles_device, tree_num_particles, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_internal_children_device, tree_num_internal_children, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_idx_first_internal_child_device, tree_idx_first_internal_child, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_total_mass_device, tree_total_mass, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_x_device, tree_center_of_mass_x, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_y_device, tree_center_of_mass_y, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_z_device, tree_center_of_mass_z, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    } 

    _acceleration_helper_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (
        a_device,
        objects_count,
        x_device,
        m_device,
        G,
        softening_length,
        opening_angle,
        width,
        leaf_morton_indices_deepest_level_device,
        sorted_indices_device,
        tree_start_particle_sorted_idx_device,
        tree_num_particles_device,
        tree_num_internal_children_device,
        tree_idx_first_internal_child_device,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    error = hipMemcpy(a, a_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_GPU_TO_CPU;
        goto err_memcpy_gpu_to_cpu;
    }

    hipFree(a_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);

    return SUCCESS;

err_memcpy_gpu_to_cpu:
err_memcpy_cpu_to_gpu:
err_gpu_memory:
    hipFree(a_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);
    return return_code;
}

extern "C"
{
    WIN32DLL_API int acceleration_barnes_hut_cuda(
        double *__restrict a,
        const System *__restrict system,
        AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;

        const int objects_count = system->objects_count;
        const double *__restrict x = system->x;
        const double *__restrict m = system->m;
        const double G = system->G;
        const double softening_length = acceleration_param->softening_length; 
        const double opening_angle = acceleration_param->opening_angle;

        /* Empty the input array */
        for (int i = 0; i < objects_count; i++)
        {
            a[i * 3 + 0] = 0.0;
            a[i * 3 + 1] = 0.0;
            a[i * 3 + 2] = 0.0;
        }

        /* Find the width and center of the bounding box */
        double center[3];
        double width;
        _calculate_bounding_box(objects_count, x, center, &width);

        /* Construct the octree */
        // Variables to be used in the octree
        int factor;
        int allocated_internal_nodes;
        int actual_num_internal_nodes;
        int *tree_start_particle_sorted_idx;
        int *tree_num_particles;
        int *tree_num_internal_children;
        int *tree_idx_first_internal_child;
        double *tree_total_mass;
        double *tree_center_of_mass_x;
        double *tree_center_of_mass_y;
        double *tree_center_of_mass_z;

        // Allocate memory
        int64 *leaf_morton_indices_deepest_level = (int64*) malloc(objects_count * sizeof(int64));
        int *sorted_indices = (int*) malloc(objects_count * sizeof(int));
        if (!leaf_morton_indices_deepest_level || !sorted_indices)
        {
            return_code = ERROR_BARNES_HUT_MORTON_INDICES_MEMORY_ALLOC;
            goto err_morton_indices_memory_alloc;
        }

        for (int i = 0; i < objects_count; i++)
        {
            sorted_indices[i] = i;
        }
        _compute_3d_morton_indices_level_21(
            leaf_morton_indices_deepest_level,
            objects_count,
            x,
            center,
            width
        );
        return_code = _radix_sort_particles_morton_index(
            objects_count,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            MORTON_MAX_LEVEL
        );
        if (return_code != SUCCESS)
        {
            goto err_radix_sort;
        }

        // Allocate memory for the octree
        factor = 1;
        if (MAX_NUM_PARTICLES_PER_LEAF <= 2)
        {
            factor = 2;
        }
        allocated_internal_nodes = factor * objects_count;

        // Start index of the particles in the node
        tree_start_particle_sorted_idx = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Number of particles in the node
        tree_num_particles = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Number of internal children of the node (i.e. not leaf)
        tree_num_internal_children = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Index to the first internal child of the node
        tree_idx_first_internal_child = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Total mass of the node
        tree_total_mass = (double*) malloc(allocated_internal_nodes * sizeof(double));

        // Center of mass of the node
        tree_center_of_mass_x = (double*) malloc(allocated_internal_nodes * sizeof(double));
        tree_center_of_mass_y = (double*) malloc(allocated_internal_nodes * sizeof(double));
        tree_center_of_mass_z = (double*) malloc(allocated_internal_nodes * sizeof(double));

        if (
            !tree_start_particle_sorted_idx ||
            !tree_num_particles ||
            !tree_num_internal_children ||
            !tree_idx_first_internal_child ||
            !tree_total_mass ||
            !tree_center_of_mass_x ||
            !tree_center_of_mass_y ||
            !tree_center_of_mass_z
        )
        {
            return_code = ERROR_BARNES_HUT_OCTREE_MEMORY_ALLOC;
            goto err_octree_memory_alloc;
        }

        tree_start_particle_sorted_idx[0] = 0;
        tree_num_particles[0] = objects_count;
        tree_num_internal_children[0] = 0;
        tree_center_of_mass_x[0] = center[0];
        tree_center_of_mass_y[0] = center[1];
        tree_center_of_mass_z[0] = center[2];

        return_code = _construct_octree(
            &allocated_internal_nodes,
            &actual_num_internal_nodes,
            x,
            m,
            width,
            sorted_indices,
            leaf_morton_indices_deepest_level,
            MORTON_MAX_LEVEL,
            &tree_start_particle_sorted_idx,
            &tree_num_particles,
            &tree_num_internal_children,
            &tree_idx_first_internal_child,
            &tree_total_mass,
            &tree_center_of_mass_x,
            &tree_center_of_mass_y,
            &tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_octree;
        }

        /* Compute the acceleration */ 
        return_code = _compute_acceleration(
            a,
            objects_count,
            x,
            m,
            G,
            softening_length,
            opening_angle,
            width,
            actual_num_internal_nodes,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            tree_start_particle_sorted_idx,
            tree_num_particles,
            tree_num_internal_children,
            tree_idx_first_internal_child,
            tree_total_mass,
            tree_center_of_mass_x,
            tree_center_of_mass_y,
            tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_acceleration;
        }

        /* Free the memory */
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);

        return SUCCESS;

    err_acceleration:
    err_octree:
    err_octree_memory_alloc:
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
    err_radix_sort:
    err_morton_indices_memory_alloc:
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return return_code;
    }
}

__global__ void memcpy_particles_array_double_to_float(
    const double *__restrict x_double,
    const double *__restrict m_double,
    const int objects_count,
    float *__restrict x,
    float *__restrict m
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    x[i * 3 + 0] = x_double[i * 3 + 0];
    x[i * 3 + 1] = x_double[i * 3 + 1];
    x[i * 3 + 2] = x_double[i * 3 + 2];
    m[i] = m_double[i];

    return;
}

__global__ void memcpy_nodes_array_double_to_float(
    const double *__restrict tree_total_mass_double,
    const double *__restrict tree_center_of_mass_x_double,
    const double *__restrict tree_center_of_mass_y_double,
    const double *__restrict tree_center_of_mass_z_double,
    const int actual_num_internal_nodes,
    float *__restrict tree_total_mass,
    float *__restrict tree_center_of_mass_x,
    float *__restrict tree_center_of_mass_y,
    float *__restrict tree_center_of_mass_z
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= actual_num_internal_nodes)
    {
        return;
    }

    tree_total_mass[i] = tree_total_mass_double[i];
    tree_center_of_mass_x[i] = tree_center_of_mass_x_double[i];
    tree_center_of_mass_y[i] = tree_center_of_mass_y_double[i];
    tree_center_of_mass_z[i] = tree_center_of_mass_z_double[i];

    return;
}

/**
 * \brief Helper kernel function for computing the acceleration of one particle in single precision
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
 __global__ void _acceleration_helper_float_kernel(
    double *__restrict a,
    const int objects_count,
    const float *__restrict x,
    const float *__restrict m,
    const float G,
    const float softening_length,
    const float opening_angle,
    const float width,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const float *__restrict tree_total_mass,
    const float *__restrict tree_center_of_mass_x,
    const float *__restrict tree_center_of_mass_y,
    const float *__restrict tree_center_of_mass_z
)
{
    typedef struct Stack
    {
        int node;
        int processed_children;
        struct Stack *last;
    } Stack;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= objects_count)
    {
        return;
    }

    const int idx_i = sorted_indices[i];
    const int64 morton_index_i = leaf_morton_indices_deepest_level[idx_i];
    const float3 x_i = make_float3(x[idx_i * 3 + 0], x[idx_i * 3 + 1], x[idx_i * 3 + 2]);
    const float softening_squared = softening_length * softening_length;

    Stack stack_pool[MORTON_MAX_LEVEL];
    Stack *stack = &(stack_pool[0]);
    stack->processed_children = -1;
    stack->last = NULL;
    stack->node = 0;
    float3 local_a = make_float3(0.0, 0.0, 0.0);

    int level = 1;

    /* Tree walk */
    while (true)
    {
        const int node = stack->node;
        for (int j = (stack->processed_children) + 1; j < tree_num_internal_children[node]; j++)
        {
            const int child_j = tree_idx_first_internal_child[node] + j;
            const int num_children_j = tree_num_internal_children[child_j];
            const int start_idx_j = tree_start_particle_sorted_idx[child_j];

            /* Leaf node */
            if (num_children_j <= 0)
            {
                const int num_particles_j = tree_num_particles[child_j];
                for (int k = 0; k < num_particles_j; k++)
                {
                    const int idx_j = sorted_indices[start_idx_j + k];
                    if (idx_i == idx_j)
                    {
                        continue;
                    }

                    // Calculate \vec{R} and its norm
                    const float3 dr = make_float3(
                        x_i.x - x[idx_j * 3 + 0],
                        x_i.y - x[idx_j * 3 + 1],
                        x_i.z - x[idx_j * 3 + 2]
                    );
                    const float R_norm = sqrt(
                        dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + softening_squared
                    );

                    // Calculate the acceleration
                    const float temp_value = G * m[idx_j] / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x;
                    local_a.y -= temp_value * dr.y;
                    local_a.z -= temp_value * dr.z;
                }

                stack->processed_children = j;
                continue;
            }

            /* Internal node */
            else
            {
                bool criteria_met = false;

                // If object i is included, then we need to traverse deeper
                const bool is_included = _check_if_included(
                    morton_index_i,
                    leaf_morton_indices_deepest_level[sorted_indices[start_idx_j]],
                    level
                );

                // Check Barnes-Hut criteria
                float3 dr;
                float norm_square;
                if (!is_included)
                {
                    dr = make_float3(
                        x_i.x - tree_center_of_mass_x[child_j],
                        x_i.y - tree_center_of_mass_y[child_j],
                        x_i.z - tree_center_of_mass_z[child_j]
                    );
                    const float width_j = width / (2 << level);
                    norm_square = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
                    if (width_j / sqrt(norm_square) < opening_angle)
                    {
                        criteria_met = true;
                    }
                }

                // Traverse deeper
                if (!criteria_met)
                {
                    Stack *new_item = &(stack_pool[level + 1]);
                    new_item->node = child_j;
                    new_item->last = stack;
                    new_item->processed_children = -1;

                    stack = new_item;
                    level++;
                    break;
                }

                else
                {
                    const float R_norm = sqrt(
                        norm_square + softening_squared
                    );

                    const float temp_value = G / (R_norm * R_norm * R_norm);
                    local_a.x -= temp_value * dr.x * tree_total_mass[child_j];
                    local_a.y -= temp_value * dr.y * tree_total_mass[child_j];
                    local_a.z -= temp_value * dr.z * tree_total_mass[child_j];

                    stack->processed_children = j;
                    continue;
                }
            }
        }

        if ((stack->processed_children + 1) >= tree_num_internal_children[stack->node])
        {
            Stack *parent = stack->last;
            if (!parent)
            {
                break;
            }            
            stack = parent;
            stack->processed_children += 1;
            level--;
        }
    }

    a[idx_i * 3 + 0] = local_a.x;
    a[idx_i * 3 + 1] = local_a.y;
    a[idx_i * 3 + 2] = local_a.z;

    return;
}

/**
 * \brief Compute the acceleration of the particles with single precision
 * 
 * \param a Array of acceleration vectors
 * \param objects_count Number of objects
 * \param x Array of position vectors
 * \param m Array of masses
 * \param G Gravitational constant
 * \param softening_length Softening length
 * \param opening_angle Opening angle
 * \param width Width of the bounding box
 * \param actual_num_internal_nodes Number of internal nodes
 * \param leaf_morton_indices_deepest_level Array of Morton indices
 * \param sorted_indices Array of sorted indices
 * \param tree_start_particle_sorted_idx Array of start index of particles in the node
 * \param tree_num_particles Array of number of particles in the node
 * \param tree_num_internal_children Array of number of internal children
 * \param tree_idx_first_internal_child Array of index of the first internal child
 * \param tree_total_mass Array of total mass of the node
 * \param tree_center_of_mass_x Array of x-coordinate of the center of mass
 * \param tree_center_of_mass_y Array of y-coordinate of the center of mass
 * \param tree_center_of_mass_z Array of z-coordinate of the center of mass
 * 
 * \retval SUCCESS if successful
 */
 IN_FILE int _compute_acceleration_float(
    double *__restrict a,
    const int objects_count,
    const double *__restrict x,
    const double *__restrict m,
    const double G,
    const double softening_length,
    const double opening_angle,
    const double width,
    const int actual_num_internal_nodes,
    const int64 *__restrict leaf_morton_indices_deepest_level,
    const int *__restrict sorted_indices,
    const int *__restrict tree_start_particle_sorted_idx,
    const int *__restrict tree_num_particles,
    const int *__restrict tree_num_internal_children,
    const int *__restrict tree_idx_first_internal_child,
    const double *__restrict tree_total_mass,
    const double *__restrict tree_center_of_mass_x,
    const double *__restrict tree_center_of_mass_y,
    const double *__restrict tree_center_of_mass_z
)
{
    int return_code;

    double *__restrict a_double_device = NULL;
    double *__restrict x_double_device = NULL;
    double *__restrict m_double_device = NULL;
    double *__restrict tree_total_mass_double_device = NULL;
    double *__restrict tree_center_of_mass_x_double_device = NULL;
    double *__restrict tree_center_of_mass_y_double_device = NULL;
    double *__restrict tree_center_of_mass_z_double_device = NULL;

    float *__restrict x_device = NULL;
    float *__restrict m_device = NULL;
    int64 *__restrict leaf_morton_indices_deepest_level_device = NULL;
    int *__restrict sorted_indices_device = NULL;
    int *__restrict tree_start_particle_sorted_idx_device = NULL;
    int *__restrict tree_num_particles_device = NULL;
    int *__restrict tree_num_internal_children_device = NULL;
    int *__restrict tree_idx_first_internal_child_device = NULL;
    float *__restrict tree_total_mass_device = NULL;
    float *__restrict tree_center_of_mass_x_device = NULL;
    float *__restrict tree_center_of_mass_y_device = NULL;
    float *__restrict tree_center_of_mass_z_device = NULL;
    hipError_t error;

    /* Allocate memory on GPU */
    error = hipMalloc((double **) &a_double_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &x_double_device, objects_count * 3 * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &m_double_device, objects_count * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_total_mass_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_x_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_y_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((double **) &tree_center_of_mass_z_double_device, actual_num_internal_nodes * sizeof(double));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMalloc((float **) &x_device, objects_count * 3 * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &m_device, objects_count * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int64 **) &leaf_morton_indices_deepest_level_device, objects_count * sizeof(int64));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &sorted_indices_device, objects_count * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_start_particle_sorted_idx_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_particles_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_num_internal_children_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((int **) &tree_idx_first_internal_child_device, actual_num_internal_nodes * sizeof(int));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_total_mass_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_x_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_y_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }
    error = hipMalloc((float **) &tree_center_of_mass_z_device, actual_num_internal_nodes * sizeof(float));
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMORY_ALLOC;
        goto err_gpu_memory;
    }

    error = hipMemcpy(x_double_device, x, objects_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(m_double_device, m, objects_count * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_total_mass_double_device, tree_total_mass, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_x_double_device, tree_center_of_mass_x, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_y_double_device, tree_center_of_mass_y, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_center_of_mass_z_double_device, tree_center_of_mass_z, actual_num_internal_nodes * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    } 

    error = hipMemcpy(leaf_morton_indices_deepest_level_device, leaf_morton_indices_deepest_level, objects_count * sizeof(int64), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(sorted_indices_device, sorted_indices, objects_count * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_start_particle_sorted_idx_device, tree_start_particle_sorted_idx, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_particles_device, tree_num_particles, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_num_internal_children_device, tree_num_internal_children, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }
    error = hipMemcpy(tree_idx_first_internal_child_device, tree_idx_first_internal_child, actual_num_internal_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_CPU_TO_GPU;
        goto err_memcpy_cpu_to_gpu;
    }

    memcpy_particles_array_double_to_float <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        x_double_device,
        m_double_device,
        objects_count,
        x_device,
        m_device
    );

    memcpy_nodes_array_double_to_float <<< (actual_num_internal_nodes + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
        tree_total_mass_double_device,
        tree_center_of_mass_x_double_device,
        tree_center_of_mass_y_double_device,
        tree_center_of_mass_z_double_device,
        actual_num_internal_nodes,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    hipFree(x_double_device);
    hipFree(m_double_device);
    hipFree(tree_total_mass_double_device);
    hipFree(tree_center_of_mass_x_double_device);
    hipFree(tree_center_of_mass_y_double_device);
    hipFree(tree_center_of_mass_z_double_device);

    _acceleration_helper_float_kernel <<< (objects_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (
        a_double_device,
        objects_count,
        x_device,
        m_device,
        G,
        softening_length,
        opening_angle,
        width,
        leaf_morton_indices_deepest_level_device,
        sorted_indices_device,
        tree_start_particle_sorted_idx_device,
        tree_num_particles_device,
        tree_num_internal_children_device,
        tree_idx_first_internal_child_device,
        tree_total_mass_device,
        tree_center_of_mass_x_device,
        tree_center_of_mass_y_device,
        tree_center_of_mass_z_device
    );

    error = hipMemcpy(a, a_double_device, objects_count * 3 * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return_code = ERROR_CUDA_BARNES_HUT_MEMCPY_GPU_TO_CPU;
        goto err_memcpy_gpu_to_cpu;
    }

    hipFree(a_double_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);

    return SUCCESS;

err_memcpy_cpu_to_gpu:
err_gpu_memory:
    hipFree(x_double_device);
    hipFree(m_double_device);
    hipFree(tree_total_mass_double_device);
    hipFree(tree_center_of_mass_x_double_device);
    hipFree(tree_center_of_mass_y_double_device);
    hipFree(tree_center_of_mass_z_double_device);
err_memcpy_gpu_to_cpu:
    hipFree(a_double_device);
    hipFree(x_device);
    hipFree(m_device);
    hipFree(leaf_morton_indices_deepest_level_device);
    hipFree(sorted_indices_device);
    hipFree(tree_start_particle_sorted_idx_device);
    hipFree(tree_num_particles_device);
    hipFree(tree_num_internal_children_device);
    hipFree(tree_idx_first_internal_child_device);
    hipFree(tree_total_mass_device);
    hipFree(tree_center_of_mass_x_device);
    hipFree(tree_center_of_mass_y_device);
    hipFree(tree_center_of_mass_z_device);
    return return_code;
}


extern "C"
{
    WIN32DLL_API int acceleration_barnes_hut_cuda_float(
        double *__restrict a,
        const System *__restrict system,
        AccelerationParam *__restrict acceleration_param
    )
    {
        int return_code;

        const int objects_count = system->objects_count;
        const double *__restrict x = system->x;
        const double *__restrict m = system->m;
        const double G = system->G;
        const double softening_length = acceleration_param->softening_length; 
        const double opening_angle = acceleration_param->opening_angle;

        /* Empty the input array */
        for (int i = 0; i < objects_count; i++)
        {
            a[i * 3 + 0] = 0.0;
            a[i * 3 + 1] = 0.0;
            a[i * 3 + 2] = 0.0;
        }

        /* Find the width and center of the bounding box */
        double center[3];
        double width;
        _calculate_bounding_box(objects_count, x, center, &width);

        /* Construct the octree */
        // Variables to be used in the octree
        int factor;
        int allocated_internal_nodes;
        int actual_num_internal_nodes;
        int *tree_start_particle_sorted_idx;
        int *tree_num_particles;
        int *tree_num_internal_children;
        int *tree_idx_first_internal_child;
        double *tree_total_mass;
        double *tree_center_of_mass_x;
        double *tree_center_of_mass_y;
        double *tree_center_of_mass_z;

        // Allocate memory
        int64 *leaf_morton_indices_deepest_level = (int64*) malloc(objects_count * sizeof(int64));
        int *sorted_indices = (int*) malloc(objects_count * sizeof(int));
        if (!leaf_morton_indices_deepest_level || !sorted_indices)
        {
            return_code = ERROR_BARNES_HUT_MORTON_INDICES_MEMORY_ALLOC;
            goto err_morton_indices_memory_alloc;
        }

        for (int i = 0; i < objects_count; i++)
        {
            sorted_indices[i] = i;
        }
        _compute_3d_morton_indices_level_21(
            leaf_morton_indices_deepest_level,
            objects_count,
            x,
            center,
            width
        );
        return_code = _radix_sort_particles_morton_index(
            objects_count,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            MORTON_MAX_LEVEL
        );
        if (return_code != SUCCESS)
        {
            goto err_radix_sort;
        }

        // Allocate memory for the octree
        factor = 1;
        if (MAX_NUM_PARTICLES_PER_LEAF <= 2)
        {
            factor = 2;
        }
        allocated_internal_nodes = factor * objects_count;

        // Start index of the particles in the node
        tree_start_particle_sorted_idx = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Number of particles in the node
        tree_num_particles = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Number of internal children of the node (i.e. not leaf)
        tree_num_internal_children = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Index to the first internal child of the node
        tree_idx_first_internal_child = (int*) malloc(allocated_internal_nodes * sizeof(int));

        // Total mass of the node
        tree_total_mass = (double*) malloc(allocated_internal_nodes * sizeof(double));

        // Center of mass of the node
        tree_center_of_mass_x = (double*) malloc(allocated_internal_nodes * sizeof(double));
        tree_center_of_mass_y = (double*) malloc(allocated_internal_nodes * sizeof(double));
        tree_center_of_mass_z = (double*) malloc(allocated_internal_nodes * sizeof(double));

        if (
            !tree_start_particle_sorted_idx ||
            !tree_num_particles ||
            !tree_num_internal_children ||
            !tree_idx_first_internal_child ||
            !tree_total_mass ||
            !tree_center_of_mass_x ||
            !tree_center_of_mass_y ||
            !tree_center_of_mass_z
        )
        {
            return_code = ERROR_BARNES_HUT_OCTREE_MEMORY_ALLOC;
            goto err_octree_memory_alloc;
        }

        tree_start_particle_sorted_idx[0] = 0;
        tree_num_particles[0] = objects_count;
        tree_num_internal_children[0] = 0;
        tree_center_of_mass_x[0] = center[0];
        tree_center_of_mass_y[0] = center[1];
        tree_center_of_mass_z[0] = center[2];

        return_code = _construct_octree(
            &allocated_internal_nodes,
            &actual_num_internal_nodes,
            x,
            m,
            width,
            sorted_indices,
            leaf_morton_indices_deepest_level,
            MORTON_MAX_LEVEL,
            &tree_start_particle_sorted_idx,
            &tree_num_particles,
            &tree_num_internal_children,
            &tree_idx_first_internal_child,
            &tree_total_mass,
            &tree_center_of_mass_x,
            &tree_center_of_mass_y,
            &tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_octree;
        }

        /* Compute the acceleration */ 
        return_code = _compute_acceleration_float(
            a,
            objects_count,
            x,
            m,
            G,
            softening_length,
            opening_angle,
            width,
            actual_num_internal_nodes,
            leaf_morton_indices_deepest_level,
            sorted_indices,
            tree_start_particle_sorted_idx,
            tree_num_particles,
            tree_num_internal_children,
            tree_idx_first_internal_child,
            tree_total_mass,
            tree_center_of_mass_x,
            tree_center_of_mass_y,
            tree_center_of_mass_z
        );
        if (return_code != SUCCESS)
        {
            goto err_acceleration;
        }

        /* Free the memory */
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);

        return SUCCESS;

    err_acceleration:
    err_octree:
    err_octree_memory_alloc:
        free(tree_start_particle_sorted_idx);
        free(tree_num_particles);
        free(tree_num_internal_children);
        free(tree_idx_first_internal_child);
        free(tree_total_mass);
        free(tree_center_of_mass_x);
        free(tree_center_of_mass_y);
        free(tree_center_of_mass_z);
    err_radix_sort:
    err_morton_indices_memory_alloc:
        free(leaf_morton_indices_deepest_level);
        free(sorted_indices);
        return return_code;
    }
}
